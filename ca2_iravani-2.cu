
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <bits/stdc++.h>
#define THREAD_NUMBER 128
#define SIZE 227

using namespace std;
__global__ void parallel_func(int *arr, int *GX, int *GY, int* d_final_arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gx_arr[SIZE*SIZE];
    int gy_arr[SIZE*SIZE];
    if(i < SIZE*SIZE){
      if((i % SIZE == 0) | (i % SIZE == SIZE - 1) | (i < SIZE) | (i > SIZE*(SIZE-1))){
          d_final_arr[i] = 0;
          gy_arr[i] = 0;
          gx_arr[i] = 0;
      }
      else{
          gx_arr[i] = arr[i]*GX[4] + arr[i+1]*GX[5] + arr[i-1]*GX[3] +
                      arr[i - SIZE]*GX[1] + arr[i - SIZE - 1]*GX[0] + arr[i - SIZE + 1]*GX[2] +
                      arr[i + SIZE]*GX[7] + arr[i + SIZE - 1]*GX[6] + arr[i + SIZE + 1]*GX[8];
          gy_arr[i] = arr[i]*GY[4] + arr[i+1]*GY[5] + arr[i-1]*GY[3] +
                      arr[i - SIZE]*GY[1] + arr[i - SIZE - 1]*GY[0] + arr[i - SIZE + 1]*GY[2] +
                      arr[i + SIZE]*GY[7] + arr[i + SIZE - 1]*GY[6] + arr[i + SIZE + 1]*GY[8];
          
      }
      d_final_arr[i] = gy_arr[i] + gx_arr[i];
    }
    return;
}

__global__ void enhanced_parallel_func(int *arr, int *GX, int *GY, int* d_final_arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < SIZE*SIZE){
      if((i % SIZE == 0) | (i % SIZE == SIZE - 1) | (i < SIZE) | (i > SIZE*(SIZE-1))){
          d_final_arr[i] = 0;
      }
      else{
          d_final_arr[i] = arr[i]*(GX[4]+GY[4]) + arr[i+1]*(GX[5]+GY[5]) + arr[i-1]*(GX[3]+GY[3]) +
                  arr[i - SIZE]*(GX[1]+GY[1]) + arr[i - SIZE - 1]*(GX[0]+GY[0]) + arr[i - SIZE + 1]*(GX[2]+GY[2]) +
                  arr[i + SIZE]*(GX[7]+GY[7]) + arr[i + SIZE - 1]*(GX[6]+GY[6]) + arr[i + SIZE + 1]*(GX[8]+GY[8]);
                      
          
      }
    }
    return;
}


void init(int* arr, int* gx_arr, int* gy_arr, int* final_arr, int* GX, int* GY){
    GX[0] = -1; GX[1] = 0; GX[2] = -1;
    GX[3] = -2; GX[4] = 0; GX[5] = 2;
    GX[6] = -1; GX[7] = 0; GX[8] = -1; 
    GY[0] = -1; GY[1] = -2; GY[2] = -1;
    GY[3] = 0;  GY[4] = 0;  GY[5] = 0;
    GY[6] = 1;  GY[7] = 2;  GY[8] = 1; 
    srand(time(NULL));
    for(int i = 0; i < (SIZE*SIZE); i++){
        int num = rand()%(5-0 + 1) + 0;
        arr[i] = num;
        gx_arr[i] = 0;
        gy_arr[i] = 0;
        final_arr[i] = 0;
    }
}

void serial(int* arr, int* gx_arr, int* gy_arr, int* final_arr, int* GX, int* GY){
    for(int i = SIZE; i < SIZE*(SIZE-1); i++){
        if((i % SIZE == 0) | (i % SIZE == SIZE - 1)){
            continue;
        }
        else{
            gx_arr[i] = arr[i]*GX[4] + arr[i+1]*GX[5] + arr[i-1]*GX[3] +
                        arr[i - SIZE]*GX[1] + arr[i - SIZE - 1]*GX[0] + arr[i - SIZE + 1]*GX[2] +
                        arr[i + SIZE]*GX[7] + arr[i + SIZE - 1]*GX[6] + arr[i + SIZE + 1]*GX[8];
            gy_arr[i] = arr[i]*GY[4] + arr[i+1]*GY[5] + arr[i-1]*GY[3] +
                        arr[i - SIZE]*GY[1] + arr[i - SIZE - 1]*GY[0] + arr[i - SIZE + 1]*GY[2] +
                        arr[i + SIZE]*GY[7] + arr[i + SIZE - 1]*GY[6] + arr[i + SIZE + 1]*GY[8];
        }
        final_arr[i] = gy_arr[i] + gx_arr[i];
    }
}

int main(){
    
    int* GX = (int*) malloc(sizeof(int) * 9);
    int* GY = (int*) malloc(sizeof(int) * 9);
    int* arr = (int*) malloc(sizeof(int) * SIZE * SIZE);
    int* gx_arr = (int*) malloc(sizeof(int) * SIZE * SIZE);
    int* gy_arr = (int*) malloc(sizeof(int) * SIZE * SIZE);
    int* final_arr = (int*) malloc(sizeof(int) * SIZE * SIZE);
    int* h_parallel = (int*) malloc(sizeof(int) * SIZE * SIZE);

    clock_t serial_start, serial_end;
    clock_t parallel_start, parallel_end;

    init(arr, gx_arr, gy_arr, final_arr, GX, GY);

    serial_start = clock();
    serial(arr, gx_arr, gy_arr, final_arr, GX, GY);
    serial_end = clock();
    cout << "serial runtime = " << serial_end - serial_start << endl;
    
    
    //cuda
    dim3 block(THREAD_NUMBER);
    dim3 grid((SIZE*SIZE + block.x - 1) / block.x);
    cout << "grid = " << grid.x << "  block = " << block.x << endl;
    int* d_arr; 
    int* d_gx; 
    int* d_gy; 
    int* d_final_arr; 

    hipMalloc((int**)&d_arr, sizeof(int) * SIZE * SIZE);
    hipMalloc((int**)&d_gx, sizeof(int) * 9);
    hipMalloc((int**)&d_gy, sizeof(int) * 9);
    hipMalloc((int**)&d_final_arr, sizeof(int) * SIZE * SIZE);

    parallel_start = clock();

    hipMemcpy(d_arr, arr, sizeof(int)*SIZE*SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_gx, GX, sizeof(int)*9, hipMemcpyHostToDevice);
    hipMemcpy(d_gy, GY, sizeof(int)*9, hipMemcpyHostToDevice);

    enhanced_parallel_func <<<grid,block>>> (d_arr, d_gx, d_gy, d_final_arr);
    hipMemcpy(h_parallel, d_final_arr, sizeof(int)*SIZE*SIZE, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    parallel_end = clock();

    int gg = 1;
    for(int i = 0; i < SIZE*SIZE ; i++){
        if(h_parallel[i] != final_arr[i]){
            gg = 0;
        }
    }
    cout << endl;
    cout << "parallel and serial --> " << gg << endl;
    cout << "parallel runtime = " << parallel_end-parallel_start << endl;

    free(GX);
    free(GY);
    free(arr);
    free(gx_arr);
    free(gy_arr);
    free(final_arr);
    hipFree(d_arr);
    hipFree(d_gx);
    hipFree(d_gy);
    hipFree(d_final_arr);
    hipFree(h_parallel);
    return 0;
}